#include "hip/hip_runtime.h"

            #include "blocks.cuh"
            #include <hip/hip_runtime.h>
            #include <stdio.h>

            extern "C" {
                void* gpu_allocate(size_t size) {
                    void* ptr = nullptr;
                    CHECK(hipMalloc(&ptr, size));
                    CHECK(hipMemset(ptr, 0, size));
                    return ptr;
                }

                void gpu_free(void* ptr) {
                    CHECK(hipFree(ptr));
                }

                void host_to_gpu(void* gpu, void* host, size_t size) {
                    CHECK(hipMemcpy(gpu, host, size, hipMemcpyHostToDevice));
                }

                void gpu_to_host(void* host, void* gpu, size_t size) {
                    CHECK(hipMemcpy(host, gpu, size, hipMemcpyDeviceToHost));
                }

                void device_synchronize() {
                    CHECK(hipDeviceSynchronize());
                }

                int fast_allclose(float* a, float* b, size_t size, float atol, float rtol) {
                    for (size_t i = 0; i < size; ++i) {
                        if (fabs(a[i] - b[i]) > atol + rtol*fabs(b[i])) {
                            return 0;
                        }
                    }
                    return 1;
                }
            }
        